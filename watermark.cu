#include "hip/hip_runtime.h"
/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
#define WINDOWS_LEAN_AND_MEAN
#define NOMINMAX
#include <windows.h>
#pragma warning(disable : 4819)
#endif

#include <Exceptions.h>
#include <ImageIO.h>
#include <ImagesCPU.h>
#include <ImagesNPP.h>

#include <assert.h>
#include <string.h>
#include <fstream>
#include <iostream>

#include <hip/hip_runtime_api.h>

#include <opencv2/core.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/imgcodecs.hpp>

static int debug_level = 1;
// #define DEBUG(level, format) DEBUG(level, format, ) 
#define DEBUG(level, format, ...) if(debug_level >= level) printf(format __VA_OPT__(,) __VA_ARGS__)

bool printfNPPinfo(int argc, char *argv[])
{
  const NppLibraryVersion *libVer = nppGetLibVersion();

  printf("NPP Library Version %d.%d.%d\n", libVer->major, libVer->minor,
         libVer->build);

  int driverVersion, runtimeVersion;
  hipDriverGetVersion(&driverVersion);
  hipRuntimeGetVersion(&runtimeVersion);

  printf("  CUDA Driver  Version: %d.%d\n", driverVersion / 1000,
         (driverVersion % 100) / 10);
  printf("  CUDA Runtime Version: %d.%d\n", runtimeVersion / 1000,
         (runtimeVersion % 100) / 10);

  // Min spec is SM 1.0 devices
  bool bVal = checkCudaCapabilities(1, 0);
  return bVal;
}

void ensureFileOpens(char const *filename){
  std::ifstream infile(filename, std::ifstream::in);

  if (infile.good())
  {
    std::cout << "boxFilterNPP opened: <" << filename
              << "> successfully!" << std::endl;
    infile.close();
  }
  else
  {
    std::cout << "boxFilterNPP unable to open: <" << filename << ">"
              << std::endl;
    infile.close();
    exit(EXIT_FAILURE);
  }
  return;
}

//__global__ void watermark_kernel(uchar *dImg, uchar *dWatermark, uchar *dAlpha, int nElements)
__global__ void watermark_kernel(uchar *dImg, uchar *dWatermark, int nPixels)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nPixels; i += blockDim.x * gridDim.x) 
      {
        // for row r, col c, channel ch:
        //   cv::Mat data field layout is array[(nchannels*mat.step*r) + (nchannels*c) + ch];
        //   See: https://stackoverflow.com/questions/37040787/opencv-in-memory-mat-representation
        // int alphaIdx = i/3; // 1 channel instead of 3. 
          // dImg[i] = (uchar) (alpha*dImg[i] + (1.0-alpha)*dImg[i]);
        //   float alpha = ((float) dAlpha[alphaIdx])/255.0;
        //   dImg[i] = (uchar) (alpha*dImg[i] + (1.0-alpha)*dWatermark[i]);
        int bIdx = 3*i;
        int gIdx = 3*i+1;
        int rIdx = 3*i+2;
        uchar bWater = dWatermark[4*i];
        uchar gWater = dWatermark[4*i+1];
        uchar rWater = dWatermark[4*i+2];
        float alpha = ((float) dWatermark[4*i+3])/255.0f; ///255.0f;
        dImg[bIdx] = (uchar) ((1.0-alpha)*dImg[bIdx] + alpha*bWater);
        dImg[gIdx] = (uchar) ((1.0-alpha)*dImg[gIdx] + alpha*gWater);
        dImg[rIdx] = (uchar) ((1.0-alpha)*dImg[rIdx] + alpha*rWater);
      }
}


void launch_watermark_kernel(cv::Mat mImg, cv::Mat mWatermark, cv::Mat &mOut)
{
  // DEBUG(1, "watermark channels: %d\n", mWatermark.channels());
  cv::Mat mWatermarkResized;
  cv::resize(mWatermark, mWatermarkResized, mImg.size(), cv::INTER_LINEAR);

  // cv::Mat mAlpha;
  // cv::extractChannel(mWatermark, mAlpha, 3);
  DEBUG(1, "watermark channels: %d\n", mWatermark.channels());
  DEBUG(1, "mImg step: %d\n", mImg.step);
  DEBUG(1, "mWatermarkResized step: %d\n", mWatermarkResized.step);

  assert(mImg.isContinuous());
  assert(mWatermark.isContinuous());

  uchar *dImg;
  uchar *dWatermark;
  // uchar *dAlpha;

  // int nImgElements = mImg.total()*mImg.elemSize();
  // int nWatermarkElements = mWatermarkResized.total()*mWatermarkResized.elemSize();
  assert(mImg.channels() == mWatermarkResized.channels()-1);
  int imgSize = mImg.total()*mImg.elemSize();
  int watermarkSize = mWatermarkResized.total()*mWatermarkResized.elemSize();
  hipError_t err = hipMalloc((void **) &dImg, imgSize);
  if(err != hipSuccess){
    fprintf(stderr, "hipMalloc dImg failed: %s\n", hipGetErrorString(err));
    std::exit(EXIT_FAILURE);
  }
  err = hipMalloc((void **) &dWatermark, watermarkSize);
  if(err != hipSuccess){
    fprintf(stderr, "hipMalloc dWatermark failed: %s\n", hipGetErrorString(err));
    std::exit(EXIT_FAILURE);
  }
  // err = hipMalloc((void **) &dAlpha, sizeAlpha);
  // if(err != hipSuccess){
  //   fprintf(stderr, "hipMalloc dAlpha failed: %s\n", hipGetErrorString(err));
  //   std::exit(EXIT_FAILURE);
  // }
  printf("mWater.step[0]=%d mWater.step[1]=%d\n", mWatermarkResized.step[0], mWatermarkResized.step[1]);
  for(int r=0; r<mWatermarkResized.rows; ++r){
    for(int c=0; c<mWatermarkResized.cols; ++c){
        for(int ch=3; ch<mWatermarkResized.channels(); ++ch){
            // printf("r c ch idx = %d %d %d %d\n", r, c, ch, (mWatermarkResized.channels()*mWatermarkResized.step[0]*r) + (mWatermarkResized.channels()*c) + ch);
            uchar val = mWatermarkResized.data[(mWatermarkResized.channels()*mWatermarkResized.step[0]*r) + (mWatermarkResized.channels()*c) + ch];
            // uchar val = 0;
             if(val > 0)
               printf("mWatermarkResized: %d\n", val);
        }
    }
  }

  err = hipMemcpy(dImg, mImg.data, imgSize, hipMemcpyHostToDevice);
  if (err != hipSuccess)
  {
      fprintf(stderr, "hipMemcpy dImg mImg Error:\n");
      fprintf(stderr, "%s\n", hipGetErrorString(err));
      std::exit(EXIT_FAILURE);
  }
  err = hipMemcpy(dWatermark, mWatermarkResized.data, watermarkSize, hipMemcpyHostToDevice);
  if (err != hipSuccess)
  {
      fprintf(stderr, "hipMemcpy dWatermark mWatermarkResized Error:\n");
      fprintf(stderr, "%s\n", hipGetErrorString(err));
      std::exit(EXIT_FAILURE);
  }
  // err = hipMemcpy(dAlpha, mAlpha.data, sizeAlpha, hipMemcpyHostToDevice);
  // if (err != hipSuccess)
  // {
  //     fprintf(stderr, "hipMemcpy dAlpha mAlpha Error:\n");
  //     fprintf(stderr, "%s\n", hipGetErrorString(err));
  //     std::exit(EXIT_FAILURE);
  // }

  int blockSize, gridSize;
  err = hipOccupancyMaxPotentialBlockSize(&gridSize, &blockSize, watermark_kernel);

  DEBUG(1, "pre kernel launch\n");
  int nPixels = mImg.total();
  watermark_kernel<<<gridSize, blockSize>>>(dImg, dWatermark, nPixels);
  // watermark_kernel<<<gridSize, blockSize>>>(dImg, dWatermark, dAlpha, nElements);
  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
      fprintf(stderr, "CUDA Runtime Error:\n");
      fprintf(stderr, "%s\n", hipGetErrorString(err));
      std::exit(EXIT_FAILURE);
  }
  DEBUG(1, "post kernel launch\n");
  uchar *data = (uchar *) malloc(imgSize);
  err = hipMemcpy(data, dImg, imgSize, hipMemcpyDeviceToHost);
  if (err != hipSuccess)
  {
      fprintf(stderr, "hipMemcpy Error:\n");
      fprintf(stderr, "%s\n", hipGetErrorString(err));
      std::exit(EXIT_FAILURE);
  }
  cv::Mat out(mImg.size(), mImg.type(), data);
  imwrite("output.jpg", out);
  imwrite("watermark.png", mWatermarkResized);

}

int main(int argc, char *argv[])
{
  printf("%s Starting...\n\n", argv[0]);

    findCudaDevice(argc, (const char **)argv);

    if (printfNPPinfo(argc, argv) == false)
    {
      exit(EXIT_SUCCESS);
    }

    if(argc != 3 && argc != 4){
      printf("Usage: %s <img-filename.png> <watermark-filename.png> [output.png]\nOutput defaults to output.png\n", argv[0]);
      exit(1);
    }

    ensureFileOpens(argv[1]);
    ensureFileOpens(argv[2]);
    std::string sImgFile = argv[1];
    std::string sWatermarkFile = argv[2];
    std::string sOutputFile = (argc == 4) ? argv[3] : "output.png";

    cv::Mat mImg = cv::imread(argv[1], cv::IMREAD_COLOR);
    // cv::Mat mWatermark = cv::imread(argv[2], cv::IMREAD_COLOR);
    cv::Mat mWatermark = cv::imread(argv[2], cv::IMREAD_UNCHANGED);

    cv::Mat mOut;
    // cv::Mat mImg = cv::imread(argv[1], cv::IMREAD_GRAYSCALE);
    // cv::Mat mWatermark = cv::imread(argv[2], cv::IMREAD_GRAYSCALE);
    launch_watermark_kernel(mImg, mWatermark, mOut);

    /*
    // declare a host image object for an 8-bit grayscale image
    npp::ImageCPU_8u_C1 oHostSrc;
    // load gray-scale image from disk
    npp::loadImage(sFilename, oHostSrc);
    // declare a device image and copy construct from the host image,
    // i.e. upload host to device
    npp::ImageNPP_8u_C1 oDeviceSrc(oHostSrc);

    // create struct with box-filter mask size
    NppiSize oMaskSize = {5, 5};

    NppiSize oSrcSize = {(int)oDeviceSrc.width(), (int)oDeviceSrc.height()};
    NppiPoint oSrcOffset = {0, 0};

    // create struct with ROI size
    NppiSize oSizeROI = {(int)oDeviceSrc.width(), (int)oDeviceSrc.height()};
    // allocate device image of appropriately reduced size
    npp::ImageNPP_8u_C1 oDeviceDst(oSizeROI.width, oSizeROI.height);
    // set anchor point inside the mask to (oMaskSize.width / 2,
    // oMaskSize.height / 2) It should round down when odd
    NppiPoint oAnchor = {oMaskSize.width / 2, oMaskSize.height / 2};

    // run box filter
    NPP_CHECK_NPP(nppiFilterBoxBorder_8u_C1R(
        oDeviceSrc.data(), oDeviceSrc.pitch(), oSrcSize, oSrcOffset,
        oDeviceDst.data(), oDeviceDst.pitch(), oSizeROI, oMaskSize, oAnchor,
        NPP_BORDER_REPLICATE));

    // declare a host image for the result
    npp::ImageCPU_8u_C1 oHostDst(oDeviceDst.size());
    // and copy the device result data into it
    oDeviceDst.copyTo(oHostDst.data(), oHostDst.pitch());

    saveImage(sResultFilename, oHostDst);
    std::cout << "Saved image: " << sResultFilename << std::endl;

    nppiFree(oDeviceSrc.data());
    nppiFree(oDeviceDst.data());

    exit(EXIT_SUCCESS);
  }
  catch (npp::Exception &rException)
  {
    std::cerr << "Program error! The following exception occurred: \n";
    std::cerr << rException << std::endl;
    std::cerr << "Aborting." << std::endl;

    exit(EXIT_FAILURE);
  }
  catch (...)
  {
    std::cerr << "Program error! An unknow type of exception occurred. \n";
    std::cerr << "Aborting." << std::endl;

    exit(EXIT_FAILURE);
    return -1;
  }
  */

  return 0;
}
